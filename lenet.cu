#include <iostream>
#include <cstdio>
#include <cstdlib>

#include "readdata.h"

// #include <hipDNN.h>
// #include <hip/hip_runtime.h>
// #include <>
// #include <hip/hip_runtime.h>
using namespace std;
int main()
{
	//braing the data and labels for training and test
	uint8_t *trainLabels, *trainImages;
	size_t trainRows, trainCols, trainSize;
	uint8_t *testLabels, *testImages;
	size_t testRows, testCols, testSize;

	if (readData("/home/roni/lenetPrac/train-images-idx3-ubyte", "/home/roni/lenetPrac/train-labels-idx1-ubyte",
			trainImages, trainLabels, trainSize, trainRows, trainCols)) {
		cerr<<"error"<<endl;
		return -1;
	}
	cout<<"Number of images in training are : "<<trainSize<<endl;
	cout<<"Number of Rows in training are : "<<trainRows<<endl;
	cout<<"Number of Cols in training are : "<<trainCols<<endl;
	if (readData("/home/roni/lenetPrac/t10k-images-idx3-ubyte", "/home/roni/lenetPrac/t10k-labels-idx1-ubyte",
			testImages, testLabels, testSize,testRows, testCols)) {
		cerr<<"error"<<endl;
		return -1;
	}
	cout<<"Number of images in testing are"<<testSize<<endl;
	cout<<"Number of Rows in testing are : "<<testRows<<endl;
	cout<<"Number of Cols in testing are : "<<testCols<<endl;
	return 0;
}