#include <iostream>
#include <cstdio>
#include <cstdlib>

#include "readdata.h"

#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
using namespace std;

struct ConvolutionLayer {
	int in_channels, out_channels, kernel_size;
	int in_hight, out_hight, in_width, out_width;
	//here stride is 1
	ConvolutionLayer(int in_channels_, int out_channels_, int kernel_size_,
				int kernel_size_, int in_hight_, int in_width_) {
		in_channels = in_channels_;
		out_channels = out_channels_;
		kernel_size = kernel_size_;
		in_hight = in_hight_;
		in_width = in_width_;
		out_hight = in_hight_ - kernel_size_ + 1;
		out_width = in_width_ - kernel_size_ + 1;
	}
};
struct PoolingLayer {
	int size, stride;
	PoolingLayer(int size_,int stride_) {
		size = size_;
		stride = stride_;
	}
};
struct FullyConnectedLayer {
	int inputs, outputs;
	FullyConnectedLayer(int inputs_, int outputs_) {
		inputs = inputs_;
		outputs = outputs_;
	}
}
struct TrainingContext {
	int batch_size;

}
int main()
{
	//braing the data and labels for training and test
	uint8_t *trainLabels, *trainImages;
	size_t trainRows, trainCols, trainSize;
	uint8_t *testLabels, *testImages;
	size_t testRows, testCols, testSize;

	if (readData("/home/roni/lenetPrac/train-images-idx3-ubyte", "/home/roni/lenetPrac/train-labels-idx1-ubyte",
			trainImages, trainLabels, trainSize, trainRows, trainCols)) {
		cerr<<"error"<<endl;
		return -1;
	}
	// cout<<"Number of images in training are : "<<trainSize<<endl;
	// cout<<"Number of Rows in training are : "<<trainRows<<endl;
	// cout<<"Number of Cols in training are : "<<trainCols<<endl;
	if (readData("/home/roni/lenetPrac/t10k-images-idx3-ubyte", "/home/roni/lenetPrac/t10k-labels-idx1-ubyte",
			testImages, testLabels, testSize,testRows, testCols)) {
		cerr<<"error"<<endl;
		return -1;
	}
	// cout<<"Number of images in testing are"<<testSize<<endl;
	// cout<<"Number of Rows in testing are : "<<testRows<<endl;
	// cout<<"Number of Cols in testing are : "<<testCols<<endl;
	ConvolutionLayer conv1(1, 20, 5, trainRows, trainCols);
	PoolingLayer pool1(2,2);
	ConvolutionLayer conv2(conv1.out_channels, 50, 5, conv1.out_hight/pool1.stride, conv2.out_width/pool1.stride);
	PoolingLayer pool2(2,2);


	return 0;
}